// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_utils.cu
 *
 * @brief Utility Routines for Tests
 */
#include <gunrock/util/test_utils.h>
#include <gunrock/util/error_utils.cuh>

namespace gunrock {
namespace util {

bool IsDevicePointer(const void *ptr)
{
    hipPointerAttribute_t attributes;
    auto err = hipPointerGetAttributes(&attributes, ptr);

    // An error here indicates the memory was LIKELY
    // allocated on the host or the pointer is gibberish.
    if(err != hipSuccess)
    {
        // Clear out the last cuda error. We expected this error
        // because it implies we have a host side pointer.
        hipGetLastError();
        return false;
    }

    if(attributes.devicePointer != nullptr)
    {
        return true;
    }

    return false;
}

/******************************************************************************
 * Device initialization
 ******************************************************************************/

void DeviceInit(CommandLineArgs &args) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "No devices supporting CUDA.\n");
    exit(1);
  }
  std::vector<int> devs;
  args.GetCmdLineArguments("device", devs);
  if (devs.size() == 0)
    for (int i = 0; i < deviceCount; i++) devs.push_back(i);
  else if (devs.size() == 1) {
    if (devs[0] < 0) {
      devs[0] = 0;
    }
    if (devs[0] > deviceCount - 1) {
      devs[0] = deviceCount - 1;
    }
  }
  for (unsigned long i = 0; i < devs.size(); i++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devs[i]);
    if (deviceProp.major < 1) {
      fprintf(stderr, "Device does not support CUDA.\n");
      exit(1);
    }
    if (!args.CheckCmdLineFlag("quiet")) {
      printf("Using device %d: %s\n", devs[i], deviceProp.name);
    }
  }
  hipSetDevice(devs[0]);
}

hipError_t SetDevice(int dev) {
  return util::GRError(hipSetDevice(dev), "hipSetDevice failed.", __FILE__,
                       __LINE__);
}

}  // namespace util
}  // namespace gunrock
