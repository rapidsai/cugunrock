// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_knn.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

// KNN includes
#include <gunrock/app/knn/knn_app.cu>
#include <gunrock/app/knn/knn_helpers.cuh>

// App and test base includes
#include <gunrock/app/test_base.cuh>

//#define KNN_TEST_DEBUG 
#ifdef KNN_TEST_DEBUG
    #define debug(a...) fprintf(stderr, a)
#else
    #define debug(a...)
#endif

using namespace gunrock;

namespace APP_NAMESPACE = app::knn;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters& parameters, VertexT v, SizeT s, ValueT val) {
    // CLI parameters
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    // Get n dimension tuplets
    std::string labels_file = parameters.Get<std::string>("labels-file");
    util::PrintMsg("Points File Input: " + labels_file, !quiet);

    hipError_t retval = hipSuccess;
    
    std::ifstream lfile(labels_file.c_str());
    if (labels_file == "" || !lfile.is_open()){
        util::PrintMsg("File cannot be open\n", !quiet);
        return retval; 
    }

    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;
    // Creating empty graph
    GraphT graph;

    // Initialization of the points array
    util::Array1D<SizeT, ValueT> points;
    //Initialization is moved to gunrock::graphio::labels::Read ... ReadLabelsStream
    //GUARD_CU(points.Allocate(n*dim, util::HOST));
    
    util::CpuTimer cpu_timer;
    cpu_timer.Start();
    // graphio::labels is setting "n" and "dim"
    retval = gunrock::graphio::labels::Read(parameters, points);
    if (retval){
        util::PrintMsg("Reading error\n");
        return retval;
    }
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // Get number of points
    SizeT n = parameters.Get<SizeT>("n");
   
    // Get dimensional of space
    SizeT dim = parameters.Get<SizeT>("dim");

    // Get number of nearest neighbors, default k = 10
    SizeT k = parameters.Get<SizeT>("k");

    if (k >= n){
        util::PrintMsg("k has to be at most n-1", !quiet);
        return retval;
    }
 
#ifdef KNN_TEST_DEBUG
    // Debug of points:
    debug("debug points\n");
    for (int i=0; i<n; ++i){
        debug("for point %d: ", i);
        for (int j=0; j<dim; ++j){
            debug("%.lf ", points[i*dim + j]);
        }
        debug("\n");
    }
#endif

    util::PrintMsg("number of points " + std::to_string(n) + ", k " + std::to_string(k), !quiet); 
    // Reference result on CPU
    SizeT* ref_knns = NULL;
    SizeT* h_knns = (SizeT*)malloc(sizeof(SizeT) * n * k);

    if (!quick) {
      // Init datastructures for reference result on GPU
      ref_knns = (SizeT*)malloc(sizeof(SizeT) * n * k);

      // If not in `quick` mode, compute CPU reference implementation
      util::PrintMsg("__________________________", !quiet);
      util::PrintMsg("______ CPU Reference _____", !quiet);

      float elapsed = app::knn::CPU_Reference<VertexT, SizeT, ValueT>(
              parameters, points, n, dim, k, ref_knns, quiet);

      util::PrintMsg("--------------------------\n Elapsed: " + 
              std::to_string(elapsed), !quiet);
      util::PrintMsg("__________________________", !quiet);
      parameters.Set("cpu-elapsed", elapsed);
    }

    std::vector<std::string> switches{"advance-mode"};

    GUARD_CU((app::Switch_Parameters(parameters, graph, switches,
        [n, dim, k, h_knns, points, ref_knns]
        (util::Parameters& parameters, GraphT& graph) {
            return app::knn::RunTests(parameters, points, graph, n, dim, k, 
                    h_knns, ref_knns, util::DEVICE);
        })));

    if (!quick) {
      delete[] ref_knns;
    }

    return retval;
  }
};

int main(int argc, char** argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test knn");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::knn::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
                           app::SIZET_U32B | app::SIZET_U64B |
                           app::VALUET_F32B | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
